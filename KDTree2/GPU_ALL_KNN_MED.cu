#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------------
..Name:  GPU_ALL_KNN_MED.cu
  Desc:  This file contains the ALL-KNN kd-tree GPU kernel

  by Shawn Brown (shawndb@cs.unc.edu)
-----------------------------------------------------------------------------*/

#ifndef _GPU_ALL_KNN_2D_MED_H_
#define _GPU_ALL_KNN_2D_MED_H_


/*---------------------------------------------------------
  Includes
---------------------------------------------------------*/

#include <stdio.h>
//#include <float.h>
#include "GPUTREE_API.h"


/*---------------------------------------------------------
  Function Definitions
---------------------------------------------------------*/

/*---------------------------------------------------------
  Name: GPU_ALL_KNN_2D_MED
  Desc: Finds the 'k' Nearest Neighbors in 
		a search set 'S' for each query point in set 'Q'
  Notes:
	1.  The search set S and query set Q are the same
		for the All-KNN search.
	2.  We need to exclude zero distance results
		Otherwise, each point will return itself as
		its own nearest neighbor
	3.  The search set S is represented by a 
		static balanced cyclical KDTree
		with one search point stored per kd-tree node
---------------------------------------------------------*/

__global__ void
GPU_ALL_KNN_2D_MED
(
	GPU_NN_Result	* qrs,		// OUT: Results of KD Nearest Neighbor Algorithm
	GPUNode_2D_MED	* kdTree,	// IN: KD Tree (Nodes)
		unsigned int	* ids,		// IN: IDs (from Indexs)
	         int          rootIdx,	// IN: index of root node in KD Tree
	    unsigned int      k			// IN: number of nearest neighbors to find
)
{
	// Per Thread Local Parameters (shared memory)
	__shared__ GPUNode_2D_MED	currNodes[KNN_THREADS_PER_BLOCK];						// Current kd-tree node
	__shared__ GPU_Search		searchStack[ALL_KNN_STACK_SIZE][KNN_THREADS_PER_BLOCK];	// Search Stack
	__shared__ GPU_NN_Result	knnHeap[KD_KNN_SIZE][KNN_THREADS_PER_BLOCK];			// 'k' NN Closest Heap
	__shared__ GPUNode_2D_MED	queryPoints[KNN_THREADS_PER_BLOCK];						// Query Point

	// Per Thread Local Parameters (registers)
	unsigned int currIdx, currAxis, currInOut, nextAxis;
	unsigned int stackTop, maxHeap, countHeap;
	float dx, dy, diff, diff2, diffDist2;
	float queryValue, splitValue;
	float dist2Heap, bestDist2;
	int tidx, width, currRow, currCol, qidx;

	// Compute Thread index
	tidx = (threadIdx.y*blockDim.x) + threadIdx.x;

	// Compute Query Index
	width = gridDim.x * blockDim.x;
	currRow = (blockIdx.y * blockDim.y) + threadIdx.y;
	currCol = (blockIdx.x * blockDim.x) + threadIdx.x;
	qidx = (currRow * width) + currCol;

	// Load current Query Point into local (fast) memory
		// Slow read from RAM into shared memory
	//queryPoints[tidx] = kdTree[currIdx];
		// BUGBUG - Had to copy componentwise to avoid kernel crash
	queryPoints[tidx].pos[0] = kdTree[qidx].pos[0];
	queryPoints[tidx].pos[1] = kdTree[qidx].pos[1];

	// Compute number of elements (in grid)
	int height = gridDim.y * blockDim.y;
	int nElems = height * width;

	// Search Stack Variables
	stackTop = 0;

	// 'k' NN Heap variables
	maxHeap   = k;			// Maximum # elements on knnHeap
	countHeap = 0;			// Current # elements on knnHeap
	dist2Heap = 0.0f;		// Max Dist of any element on heap
	bestDist2 = 3.0e38f;

	// Put root search info on stack
	searchStack[stackTop][tidx].nodeFlags = (rootIdx & 0x1FFFFFFF); // | ((currAxis << 29) & 0x60000000); // | ((currInOut << 31) & 0x8000000);;
	searchStack[stackTop][tidx].splitVal  = 3.0e+38F;
	stackTop++;

	while (stackTop != 0)
	{
		// Statistics
		//best.cNodes++;

		// Get Current Node from top of stack
		stackTop--;

		// Get Node Info
		currIdx   = (searchStack[stackTop][tidx].nodeFlags & 0x1FFFFFFFU);
		currAxis  = (searchStack[stackTop][tidx].nodeFlags & 0x60000000U) >> 29;
		currInOut = (searchStack[stackTop][tidx].nodeFlags & 0x80000000U) >> 31;
		
		nextAxis  = ((currAxis == 0) ? 1 : 0);

		// Early Exit Check
		if (currInOut == 1)	// KD_OUT
		{
			if (countHeap == maxHeap) // Is heap full yet ?!?
			{
				// Next Line is effectively queryValue = queryPoints[prevAxis];
				queryValue = ((currAxis == 0) ? queryPoints[tidx].pos[1] : queryPoints[tidx].pos[0]);
				splitValue = searchStack[stackTop][tidx].splitVal;	// Split Value of Parent Node
				diff  = splitValue - queryValue;
				diff2 = diff*diff;
				if (diff2 >= dist2Heap)
				{
					// We can do an early exit for this node
					continue;
				}
			}
		}

		// WARNING - It's Much faster to load this node from global memory after the "Early Exit check" !!!

		// Load current node
		// currNodes[tidx] = kdtree[currIdx];
			// Slow read from RAM into 
			// BUGBUG - Had to copy componentwise to avoid kernel crash
		currNodes[tidx].pos[0] = kdTree[currIdx].pos[0];
		currNodes[tidx].pos[1] = kdTree[currIdx].pos[1];
		currNodes[tidx].Left   = kdTree[currIdx].Left;
		currNodes[tidx].Right  = kdTree[currIdx].Right;

		// Get Best Fit Dist for checking child ranges
		queryValue = ((currAxis == 0) ? queryPoints[tidx].pos[0] : queryPoints[tidx].pos[1]);
		splitValue = ((currAxis == 0) ? currNodes[tidx].pos[0] : currNodes[tidx].pos[1]);
		diff  = splitValue - queryValue;
		diff2 = diff*diff;

		// Calc Dist from Median Node to queryLocation
		dx = currNodes[tidx].pos[0] - queryPoints[tidx].pos[0];
		dy = currNodes[tidx].pos[1] - queryPoints[tidx].pos[1];
		diffDist2 = (dx*dx) + (dy*dy);

		// See if we should add this point to the 'k' NN Heap
		if (diffDist2 <= 0.0f)
		{
			// Do nothing, The query point found itself in the kd-tree
			// We don't want to add ourselves as a NN.
		}
		else if (countHeap < maxHeap)
		{
			//-------------------------------
			//	< 'k' elements on heap
			//	Do Simple Array append
			//-------------------------------

			countHeap++;
			knnHeap[countHeap][tidx].Id  = currIdx;
			knnHeap[countHeap][tidx].Dist = diffDist2;

			// Do we need to convert the array into a max distance heap ?!?
			if (countHeap == maxHeap)
			{
				// Yes, turn array into a heap, takes O(k) time
				for (unsigned int z = countHeap/2; z >= 1; z--)
				{
					//
					// Demote each element in turn (to correct position in heap)
					//

					unsigned int parentHIdx = z;		// Start at specified element
					unsigned int childHIdx  = z << 1;	// left child of parent

					// Compare Parent to it's children
					while (childHIdx <= maxHeap)
					{
						// Update Distances
						float parentD2 = knnHeap[parentHIdx][tidx].Dist;
						float childD2  = knnHeap[childHIdx][tidx].Dist;

						// Find largest child 
						if (childHIdx < maxHeap)
						{
							float rightD2 = knnHeap[childHIdx+1][tidx].Dist;
							if (childD2 < rightD2)
							{
								// Use right child
								childHIdx++;	
								childD2 = rightD2;
							}
						}

						// Compare largest child to parent
						if (parentD2 >= childD2) 
						{
							// Parent is larger than both children, exit loop
							break;
						}

						// Demote parent by swapping with it's largest child
						GPU_NN_Result closeTemp = knnHeap[parentHIdx][tidx];
						knnHeap[parentHIdx][tidx]    = knnHeap[childHIdx][tidx];
						knnHeap[childHIdx][tidx]     = closeTemp;
						
						// Update indices
						parentHIdx = childHIdx;	
						childHIdx  = parentHIdx<<1;		// left child of parent
					}
				}

				// Update trim distances
				dist2Heap = knnHeap[1][tidx].Dist;
				bestDist2 = dist2Heap;
			}
		}
		else if (diffDist2 < dist2Heap)
		{
			//-------------------------------
			// >= k elements on heap
			// Do Heap Replacement
			//-------------------------------

			// Replace Root Element with new element
			knnHeap[1][tidx].Id  = currIdx;
			knnHeap[1][tidx].Dist = diffDist2;

			//
			// Demote new element (to correct position in heap)
			//
			unsigned int parentHIdx = 1;	// Start at Root
			unsigned int childHIdx  = 2;	// left child of parent

			// Compare current index to it's children
			while (childHIdx <= maxHeap)
			{
				// Update Distances
				float parentD2 = knnHeap[parentHIdx][tidx].Dist;
				float childD2  = knnHeap[childHIdx][tidx].Dist;

				// Find largest child 
				if (childHIdx < maxHeap)
				{
					float rightD2 = knnHeap[childHIdx+1][tidx].Dist;
					if (childD2 < rightD2)
					{
						// Use right child
						childHIdx++;	
						childD2 = rightD2;
					}
				}

				// Compare largest child to parent
				if (parentD2 >= childD2) 
				{
					// Parent node is larger than both children, exit
					break;
				}

				// Demote parent by swapping with it's largest child
				GPU_NN_Result closeTemp = knnHeap[parentHIdx][tidx];
				knnHeap[parentHIdx][tidx]    = knnHeap[childHIdx][tidx];
				knnHeap[childHIdx][tidx]     = closeTemp;
				
				// Update indices
				parentHIdx = childHIdx;	
				childHIdx  = parentHIdx<<1;		// left child of parent
			}

			// Update Trim distances
			dist2Heap = knnHeap[1][tidx].Dist;
			bestDist2 = dist2Heap;
		}

		// update bestDist2

		if (queryValue <= splitValue)
		{
			// [...QL...BD]...SV		-> Include Left range only
			//		or
			// [...QL...SV...BD]		-> Include Both Left and Right Sub Ranges
			
			// Check if we should add Right Sub-range to stack
			if (diff2 < bestDist2)
			{
				//nextIdx = currNodes[tidx].Right;
				if (0xFFFFFFFF != currNodes[tidx].Right)	// cInvalid
				{
					// Push Onto top of stack
					searchStack[stackTop][tidx].nodeFlags = (currNodes[tidx].Right & 0x1FFFFFFFU) | ((nextAxis << 29) & 0x60000000U) | 0x80000000U;
					searchStack[stackTop][tidx].splitVal  = splitValue;
					stackTop++;
				}
			}

			// Always Add Left Sub-range to search path
			//nextIdx = currNodes[tidx].Left;
			if (0xFFFFFFFF != currNodes[tidx].Left)
			{
				// Push Onto top of stack
				searchStack[stackTop][tidx].nodeFlags = (currNodes[tidx].Left & 0x1FFFFFFFU) | ((nextAxis << 29) & 0x60000000U); // | 0x80000000U;
				searchStack[stackTop][tidx].splitVal  = splitValue;
				stackTop++;
			}

		}
		else
		{
			// SV...[BD...QL...]		-> Include Right sub range only
			//		  or
			// [BD...SV...QL...]		-> Include Both Left and Right Sub Ranges

			// Check if we should add left sub-range to search path
			if (diff2 < bestDist2)
			{
				// Add to search stack
				//nextIdx = currNodes[tidx].Left;
				if (0xFFFFFFFFU != currNodes[tidx].Left)
				{
					// Push Onto top of stack
					searchStack[stackTop][tidx].nodeFlags = (currNodes[tidx].Left & 0x1FFFFFFFU) | ((nextAxis << 29) & 0x60000000U) | 0x80000000U;
					searchStack[stackTop][tidx].splitVal  = splitValue;
					stackTop++;
				}
			}
				
			// Always Add Right Sub-range
			//nextIdx = currNodes[tidx].Right;
			if (0xFFFFFFFFU != currNodes[tidx].Right)
			{
				// Push Onto top of stack
				searchStack[stackTop][tidx].nodeFlags = (currNodes[tidx].Right & 0x1FFFFFFFU) | ((nextAxis << 29) & 0x60000000U); // | 0x8000000U;
				searchStack[stackTop][tidx].splitVal  = splitValue;
				stackTop++;
			}
		}
	}

	//
	//	Output Results
	//

	// We now have a heap of the 'k' nearest neighbors
	// Write them to the results array
	// Assume answers should be stored along z axis of 3 dimensional cube
	for (unsigned int i = 0; i < countHeap; i++)
	{
		unsigned int i1 = i+1;
		unsigned int offset = i * nElems;

		// Convert Nearest Neighbor Info to final format
			// Slow read from RAM memory
		knnHeap[i1][tidx].Id  = ids[knnHeap[i1][tidx].Id];			// Really need ID's not indexs		
		knnHeap[i1][tidx].Dist = sqrtf( knnHeap[i1][tidx].Dist );		// Get True distance (not distance squared)

		// Store Result 
			// Slow write to RAM memory
		//qrs[qidx+offset] = knnHeap[i1][tidx];
			// BUGBUG - Had to copy componentwise to avoid kernel crash
		qrs[qidx+offset].Id  = knnHeap[i1][tidx].Id;
		qrs[qidx+offset].Dist = knnHeap[i1][tidx].Dist;
	}
}

#endif // _GPU_ALL_KNN_2D_MED_H_
