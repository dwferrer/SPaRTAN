#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gsl/gsl_rng.h>

//shamelessly stolen from cuda gems nbody code
#define EPS2 0.00000001
#define NThreads 1024
__device__ float4
bodyBodyInteraction(float4 bi, float4 bj, float4 ai)
{
        float3 r;
        // r_ij [3 FLOPS]
        r.x = bj.x - bi.x;
        r.y = bj.y - bi.y;
        r.z = bj.z - bi.z;
        // distSqr = dot(r_ij, r_ij) + EPS^2 [6 FLOPS]
        float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + ai.w;
        // invDistCube =1/distSqr^(3/2) [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
        float distSixth = distSqr * distSqr * distSqr;
        float invDistCube = 1.0f/sqrtf(distSixth);
        // s = m_j * invDistCube [1 FLOP]
        float s = bj.w * invDistCube;
        // a_i = a_i + s * r_ij [6 FLOPS]
        ai.x += r.x * s;
        ai.y += r.y * s;
        ai.z += r.z * s;
        return ai;
}


template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};


__device__ float4
tile_calculation(float4 myPosition, float4 accel){
        long long  int i;
        float4 *shPosition = SharedMemory<float4>();
        #pragma unroll 32
        for (i = 0; i < blockDim.x; i++) {
        accel = bodyBodyInteraction(myPosition, shPosition[i], accel);
        }
        return -accel;
        }

__global__ void
calculate_forces(void *devX, void *devA,long long int N)
{
        float4 *shPosition = SharedMemory<float4>();
        float4 *globalX = (float4 *)devX;
        float4 *globalA = (float4 *)devA;
        float4 myPosition;
        int i, tile;
        float4 acc;
        int gtid = blockIdx.x * blockDim.x + threadIdx.x;
        myPosition = globalX[gtid];
        acc.x = globalA[gtid].x; acc.y = globalA[gtid].y; acc.z = globalA[gtid].z; acc.w = globalA[gtid].w;
        for (i = 0, tile = 0; i < N; i += NThreads, tile++) {
                int idx = tile * blockDim.x + threadIdx.x;
                shPosition[threadIdx.x] = globalX[idx];
                __syncthreads();
                acc = tile_calculation(myPosition, acc);
                __syncthreads();
        }
        // Save the result in global memory for the integration step.
        float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
        globalA[gtid] = acc4;
}


void gpugravity(float * pos, float *accel, long long int N){
        float4 *positions = (float4 *) pos;
        float4 *acc = (float4 *) accel;
        int size = N*sizeof(float4);

        float4 * d_pos, *d_acc;
        int d_size = N*sizeof(float4);
        hipMalloc((void **) &d_pos,d_size);
        hipMalloc((void **) &d_acc,d_size);


        hipMemcpy(d_pos,positions,size,hipMemcpyHostToDevice);
        hipMemcpy(d_acc,positions,size,hipMemcpyHostToDevice);

        calculate_forces<<<(N+NThreads-1)/NThreads,NThreads,NThreads*sizeof(float4)>>>(d_pos,d_acc,N);
        hipMemcpy(acc,d_acc,size,hipMemcpyDeviceToHost);
}
