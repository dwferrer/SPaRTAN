#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gsl/gsl_rng.h>

//shamelessly stolen from cuda gems nbody code
#define EPS2 0.00000001
#define NThreads 1024
__device__ float4
bodyBodyInteraction(float4 bi, float4 bj, float4 ai)
{
        float3 r;
        // r_ij [3 FLOPS]
        r.x = bj.x - bi.x;
        r.y = bj.y - bi.y;
        r.z = bj.z - bi.z;
        // distSqr = dot(r_ij, r_ij) + EPS^2 [6 FLOPS]
        float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + ai.w;
        // invDistCube =1/distSqr^(3/2) [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
        float distSixth = distSqr * distSqr * distSqr;
        float invDistCube = 1.0f/sqrtf(distSixth);
        // s = m_j * invDistCube [1 FLOP]
        float s = bj.w * invDistCube;
        // a_i = a_i + s * r_ij [6 FLOPS]
        ai.x += r.x * s;
        ai.y += r.y * s;
        ai.z += r.z * s;
        return ai;
}


template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};


__device__ float4
tile_calculation(float4 myPosition, float4 accel){
        long long  int i;
        float4 *shPosition = SharedMemory<float4>();
        #pragma unroll 32
        for (i = 0; i < blockDim.x; i++) {
        accel = bodyBodyInteraction(myPosition, shPosition[i], accel);
        }
        return accel;
        }

__global__ void
calculate_forces(void *devXsource, void * devXsink, void *devA, int Nsource, int Nsink, int numdevs)
{
        float4 *shPosition = SharedMemory<float4>();
        float4 *globalXsource = (float4 *)devXsource;
        float4 *globalXsink = (float4 *)devXsink;
        float4 *globalA = (float4 *)devA;
        float4 myPosition;
        int i, tile;
        float4 acc;
        int gtid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gtid >= Nsink) return;
        myPosition = globalXsink[gtid];
        acc.x = globalA[gtid].x; acc.y = globalA[gtid].y; acc.z = globalA[gtid].z; acc.w = globalA[gtid].w;
        for (i = 0, tile = 0; i < Nsource; i += NThreads, tile++) {
                int idx = tile * blockDim.x + threadIdx.x;
                shPosition[threadIdx.x] = globalXsource[idx];
                __syncthreads();
                acc = tile_calculation(myPosition, acc);
                __syncthreads();
        }
        // Save the result in global memory for the integration step.
        float4 acc4 = {acc.x, acc.y, acc.z, acc.w};
        globalA[gtid] = acc4;
}


#include <cassert>
#include <stdio.h>

void gpugravity(float * pos, float *accel, long long int N){
        float4 *positions = (float4 *) pos;
        float4 *acc = (float4 *) accel;
        
        int numdevs = 0;
        hipGetDeviceCount(&numdevs);
        hipStream_t * streams = new hipStream_t[numdevs];
        hipEvent_t * events = new hipEvent_t[numdevs];
        
        int * devicesinks = new int[numdevs];
        size_t *offset = new size_t[numdevs];
        size_t total_offset = 0;
        int remainingsinks = N;
        int allotment = N/numdevs;
        int d_sourcesize = N*sizeof(float4);
        
        float4 ** d_pos = new float4 *[numdevs];
        float4 ** d_acc = new float4 *[numdevs];
        
        for(int i = 0; i < numdevs; i++){
        	
        	//create the streams and events
        	hipStreamCreate(&streams[i]);
        	hipEventCreate(&events[i]);
        	
        	//figure out how many sinks to give each device
        	if (remainingsinks > allotment) devicesinks[i] = allotment;
        	else devicesinks[i] = allotment;
        	remainingsinks -= devicesinks[i];
        	
        	//calculate the offset for each device
        	
        	offset[i] = total_offset;
        	total_offset += devicesinks[i];
        	
		printf("Device %d has %d sinks and an offset of %d\n There are %d particles remaining\n\n",i,devicesinks[i],offset[i],remainingsinks);
        	
		hipSetDevice(i);
        	
        	int d_sinksize = devicesinks[i] *sizeof(float4);
        	hipMalloc((void **) &d_pos[i],d_sourcesize);
        	hipMalloc((void **) &d_acc[i],d_sinksize);      	
        }
        
        
        
        for(int i = 0; i < numdevs; i++){
        	hipSetDevice(i);
        	int d_sinksize = devicesinks[i] *sizeof(float4);
        	hipMemcpyAsync(d_pos[i], positions, d_sourcesize,hipMemcpyHostToDevice,streams[i]);
        	hipMemcpyAsync(d_acc[i], &(acc[offset[i]]), d_sinksize,hipMemcpyHostToDevice,streams[i]);    	
        }
        
        for(int i = 0; i < numdevs; i++){
        	hipSetDevice(i);
        	calculate_forces<<<(devicesinks[i]+NThreads-1)/NThreads,NThreads,NThreads*sizeof(float4),streams[i]>>>(d_pos[i],&((d_pos[i])[offset[i]]), d_acc[i] ,N, devicesinks[i],numdevs);
        }
        
        for(int i = 0; i < numdevs; i++){
        	hipSetDevice(i);
        	int d_sinksize = devicesinks[i] *sizeof(float4);
        	hipMemcpyAsync(&(acc[offset[i]]),d_acc[i],d_sinksize,hipMemcpyDeviceToHost,streams[i]);
        	hipEventRecord(events[i],streams[i]);        	
        }
        
        //wait for all devices to complete
        for(int i = 0; i < numdevs; i++) hipEventSynchronize(events[i]);    
	for(int i = 0; i < numdevs; i++){
		hipSetDevice(i); hipFree(d_pos[i]); hipFree(d_acc[i]);
	}
	delete[] offset; delete[] devicesinks;
	
}
